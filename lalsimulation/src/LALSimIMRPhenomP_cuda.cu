#include "hip/hip_runtime.h"
//==============================================================================
//
// This code was developed as part of the Astronomy Data and Computing Services
// (ADACS; https://adacs.org.au) 2017B Software Support program.
//
// Written by: Gregory B. Poole
// Date:       September 2017
//
// It is distributed under the MIT (Expat) License (see https://opensource.org/):
//
// Copyright (c) 2017 Astronomy Data and Computing Services (ADACS)
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
//
//==============================================================================

#ifdef __GNUC__
#define UNUSED __attribute__ ((unused))
#else
#define UNUSED
#endif

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <gsl/gsl_errno.h>
#include <gsl/gsl_spline.h>
#include <gsl/gsl_math.h>
#include <gsl/gsl_sf_trig.h>

#include <lal/Date.h>
#include <lal/FrequencySeries.h>
#include <lal/LALAtomicDatatypes.h>
#include <lal/LALConstants.h>
#include <lal/LALDatatypes.h>
#include <lal/LALSimInspiral.h>
#include <lal/Units.h>
#include <lal/XLALError.h>
#include <lal/SphericalHarmonics.h>
#include <lal/Sequence.h>
#include <lal/LALStdlib.h>
#include <lal/LALStddef.h>

#include "LALSimIMR.h"
// This is ugly, but allows us to reuse internal IMRPhenomC and IMRPhenomD functions without making those functions XLAL 
/*
#include "LALSimIMRPhenomC_internals.c"
#include "LALSimIMRPhenomD_internals.c"

#include "LALSimIMRPhenomP.h"
*/
#include "LALSimIMRPhenomP_cuda.h"

#include "PhenomPCoreOneFrequency.c"

#ifndef _OPENMP
#define omp ignore
#endif

// Macro functions to rotate the components of a vector about an axis 
#define ROTATEZ(angle, vx, vy, vz)\
tmp1 = vx*cos(angle) - vy*sin(angle);\
tmp2 = vx*sin(angle) + vy*cos(angle);\
vx = tmp1;\
vy = tmp2

#define ROTATEY(angle, vx, vy, vz)\
tmp1 = vx*cos(angle) + vz*sin(angle);\
tmp2 = - vx*sin(angle) + vz*cos(angle);\
vx = tmp1;\
vz = tmp2

// These functions deal with any GPU exceptions, but should be called with the macros defined in the corresponding .hh file
__host__ void _throw_on_generic_error(bool check_failure,int implementation_code, const std::string file, const std::string func, int line)
{
  if(check_failure) throw(lalsimulation_cuda_exception(GENERIC_CUDA_ERROR_CODE,implementation_code,file,func,line));
}
__host__ void _throw_on_cuda_error(hipError_t cuda_code, int implementation_code, const std::string file, const std::string func, int line)
{
  if(cuda_code != hipSuccess) throw(lalsimulation_cuda_exception((int)cuda_code,implementation_code,file,func,line));
}
__host__ void _check_for_cuda_error(int implementation_code,const std::string file, const std::string func, int line)
{
  try{
    hipError_t cuda_code = hipPeekAtLastError();
    if(cuda_code != hipSuccess)
        throw(lalsimulation_cuda_exception((int)cuda_code,implementation_code,"CUDA error detected after ",file,func,line));
  }
  catch(const lalsimulation_cuda_exception e){
      e.process_exception();
  }
}
__host__ void _check_thread_sync(int implementation_code,const std::string file, const std::string func, int line)
{
  try{
    hipError_t cuda_code = hipDeviceSynchronize();
    if(cuda_code != hipSuccess)
        throw(lalsimulation_cuda_exception((int)cuda_code,implementation_code,"Threads not synchronised after ",file,func,line));
  }
  catch(const lalsimulation_cuda_exception e){
      e.process_exception();
  }
}

__host__
void PhenomPCoreAllFrequencies_cuda(UINT4 L_fCut,
        REAL8Sequence *freqs,
        UINT4 offset,
        const REAL8 eta,
        const REAL8 chi1_l,
        const REAL8 chi2_l,
        const REAL8 chip,
        const REAL8 distance,
        const REAL8 M,
        const REAL8 phic,
        IMRPhenomDAmplitudeCoefficients *pAmp_host,
        IMRPhenomDPhaseCoefficients *pPhi_host,
        BBHPhenomCParams *PCparams_host,
        PNPhasingSeries *pn_host,
        NNLOanglecoeffs *angcoeffs_host,
        SpinWeightedSphericalHarmonic_l2 *Y2m_host,
        const REAL8 alphaNNLOoffset,
        const REAL8 alpha0,
        const REAL8 epsilonNNLOoffset,
        IMRPhenomP_version_type IMRPhenomP_version,
        AmpInsPrefactors *amp_prefactors_host,
        PhiInsPrefactors *phi_prefactors_host,
        COMPLEX16FrequencySeries *hptilde_host,
        COMPLEX16FrequencySeries *hctilde_host,
        REAL8 *phis_host,
        int   *errcode){
fprintf(stderr,"Entered cuda code.\n");

  // Copy inputs to device
  IMRPhenomDAmplitudeCoefficients pAmp;
  IMRPhenomDPhaseCoefficients pPhi;
  BBHPhenomCParams PCparams;
  PNPhasingSeries pn;
  NNLOanglecoeffs angcoeffs;
  SpinWeightedSphericalHarmonic_l2 Y2m;
  AmpInsPrefactors amp_prefactors;
  PhiInsPrefactors phi_prefactors;
  COMPLEX16FrequencySeries *hptilde;
  COMPLEX16FrequencySeries *hctilde;
  REAL8 *phis;
  try{
      throw_on_cuda_error(hipMemcpy(&pAmp,          pAmp_host,          sizeof(IMRPhenomDAmplitudeCoefficients), hipMemcpyHostToDevice),lalsimulation_cuda_exception::MEMCPY);
      throw_on_cuda_error(hipMemcpy(&pPhi,          pPhi_host,          sizeof(IMRPhenomDPhaseCoefficients),     hipMemcpyHostToDevice),lalsimulation_cuda_exception::MEMCPY);
      throw_on_cuda_error(hipMemcpy(&PCparams,      PCparams_host,      sizeof(BBHPhenomCParams),                hipMemcpyHostToDevice),lalsimulation_cuda_exception::MEMCPY);
      throw_on_cuda_error(hipMemcpy(&pn,            pn_host,            sizeof(PNPhasingSeries),                 hipMemcpyHostToDevice),lalsimulation_cuda_exception::MEMCPY);
      throw_on_cuda_error(hipMemcpy(&angcoeffs,     angcoeffs_host,     sizeof(NNLOanglecoeffs),                 hipMemcpyHostToDevice),lalsimulation_cuda_exception::MEMCPY);
      throw_on_cuda_error(hipMemcpy(&Y2m,           Y2m_host,           sizeof(SpinWeightedSphericalHarmonic_l2),hipMemcpyHostToDevice),lalsimulation_cuda_exception::MEMCPY);
      throw_on_cuda_error(hipMemcpy(&amp_prefactors,amp_prefactors_host,sizeof(AmpInsPrefactors),                hipMemcpyHostToDevice),lalsimulation_cuda_exception::MEMCPY);
      throw_on_cuda_error(hipMemcpy(&phi_prefactors,phi_prefactors_host,sizeof(PhiInsPrefactors),                hipMemcpyHostToDevice),lalsimulation_cuda_exception::MEMCPY);
      throw_on_cuda_error(hipMalloc((void **)&hptilde,L_fCut*sizeof(COMPLEX16)),lalsimulation_cuda_exception::MALLOC);
      throw_on_cuda_error(hipMalloc((void **)&hctilde,L_fCut*sizeof(COMPLEX16)),lalsimulation_cuda_exception::MALLOC);
      throw_on_cuda_error(hipMalloc((void **)&phis,   L_fCut*sizeof(REAL8)),    lalsimulation_cuda_exception::MALLOC);
  }
  catch(const lalsimulation_cuda_exception e){
      e.process_exception();
  }

  // Run kernel
  try{
    int n_threads=256;
    int grid_size=(L_fCut+(n_threads-1))/n_threads;
    PhenomPCoreOneFrequency_cuda<<<grid_size,n_threads>>>(L_fCut,
          freqs,
          offset,
          eta,
          chi1_l,
          chi2_l,
          chip,
          distance,
          M,
          phic,
          &pAmp,
          &pPhi,
          &PCparams,
          &pn,
          &angcoeffs,
          &Y2m,
          alphaNNLOoffset,
          alpha0,
          epsilonNNLOoffset,
          IMRPhenomP_version,
          &amp_prefactors,
          &phi_prefactors,
          hptilde,
          hctilde,
          phis);
  }
  // Alter this to return an error code on kernel errorcode exception
  catch(const lalsimulation_cuda_exception e){
      e.process_exception();
  }

  // Offload results
  try{
      throw_on_cuda_error(hipMemcpy(hptilde_host,hptilde,L_fCut*sizeof(COMPLEX16FrequencySeries),hipMemcpyDeviceToHost),lalsimulation_cuda_exception::MEMCPY);
      throw_on_cuda_error(hipMemcpy(hctilde_host,hctilde,L_fCut*sizeof(COMPLEX16FrequencySeries),hipMemcpyDeviceToHost),lalsimulation_cuda_exception::MEMCPY);
      throw_on_cuda_error(hipMemcpy(phis_host,   phis,   L_fCut*sizeof(REAL8),                   hipMemcpyDeviceToHost),lalsimulation_cuda_exception::MEMCPY);
  }
  catch(const lalsimulation_cuda_exception e){
      e.process_exception();
  }

  // Clean-up
  try{
      throw_on_cuda_error(hipFree(hptilde),lalsimulation_cuda_exception::FREE);
      throw_on_cuda_error(hipFree(hctilde),lalsimulation_cuda_exception::FREE);
      throw_on_cuda_error(hipFree(phis),   lalsimulation_cuda_exception::FREE);
  }
  catch(const lalsimulation_cuda_exception e){
      e.process_exception();
  }
fprintf(stderr,"Leaving cuda code.\n");
}

__global__
void PhenomPCoreOneFrequency_cuda(UINT4 L_fCut,
        REAL8Sequence *freqs,
        UINT4 offset,
        const REAL8 eta,
        const REAL8 chi1_l,
        const REAL8 chi2_l,
        const REAL8 chip,
        const REAL8 distance,
        const REAL8 M,
        const REAL8 phic,
        IMRPhenomDAmplitudeCoefficients *pAmp,
        IMRPhenomDPhaseCoefficients *pPhi,
        BBHPhenomCParams *PCparams,
        PNPhasingSeries *pn,
        NNLOanglecoeffs *angcoeffs,
        SpinWeightedSphericalHarmonic_l2 *Y2m,
        const REAL8 alphaNNLOoffset,
        const REAL8 alpha0,
        const REAL8 epsilonNNLOoffset,
        IMRPhenomP_version_type IMRPhenomP_version,
        AmpInsPrefactors *amp_prefactors,
        PhiInsPrefactors *phi_prefactors,
        COMPLEX16FrequencySeries *hptilde,
        COMPLEX16FrequencySeries *hctilde,
        REAL8 *phis){

    UINT4 i = (UINT4)(blockIdx.x*blockDim.x + threadIdx.x);
    if(i < L_fCut){

      COMPLEX16 hp_val;
      COMPLEX16 hc_val;
      REAL8     phasing;
      double    f = freqs->data[i];
      int       j = i + offset; // shift index for frequency series if needed

      // Generate the waveform 
      int per_thread_errcode = PhenomPCoreOneFrequency(f,
                                                       eta, chi1_l, chi2_l, chip, distance, M, phic,
                                                       pAmp, pPhi, PCparams, pn, angcoeffs, Y2m,
                                                       alphaNNLOoffset - alpha0, epsilonNNLOoffset,
                                                       &hp_val, &hc_val, &phasing, IMRPhenomP_version, amp_prefactors, phi_prefactors);

      // THROW EXCEPTION HERE INSTEAD
      //if (per_thread_errcode != XLAL_SUCCESS) {
      //  (*errcode) = per_thread_errcode;
      //}

      (hptilde->data->data)[j] = hp_val;
      (hctilde->data->data)[j] = hc_val;

      phis[i] = phasing;

  }
}
