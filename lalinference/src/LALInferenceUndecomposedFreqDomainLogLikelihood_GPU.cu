#include "hip/hip_runtime.h"
/*
*  Copyright (C) 2011 Alex Ayerdi
*
*  This program is free software; you can redistribute it and/or modify
*  it under the terms of the GNU General Public License as published by
*  the Free Software Foundation; either version 2 of the License, or
*  (at your option) any later version.
*
*  This program is distributed in the hope that it will be useful,
*  but WITHOUT ANY WARRANTY; without even the implied warranty of
*  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*  GNU General Public License for more details.
*
*  You should have received a copy of the GNU General Public License
*  along with with program; see the file COPYING. If not, write to the
*  Free Software Foundation, Inc., 59 Temple Place, Suite 330, Boston,
*  MA  02111-1307  USA
*/

/*-----------------------------------------------------------------------
 *
 * File Name: LALInferenceUndecomposedFreqDomainLogLikelihood_GPU.cu
 *
 * Author: Ayerdi, A.
 *
 *
 *-----------------------------------------------------------------------
 */

#include <stdio.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#include "LALInferenceUndecomposedFreqDomainLogLikelihood_GPU.h"

#define MAX_THREADS 512

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }                         
}

__global__ void chisquared_LogLikelihood_Kernel(REAL8 *d_sum, int lower, int dataSize,
						COMPLEX16 *freqModelhPlus_Data,
						COMPLEX16 *freqModelhCross_Data,
						COMPLEX16 *freqData_Data,
						REAL8 *oneSidedNoisePowerSpectrum_Data,
						double FplusScaled,
						double FcrossScaled,
						double deltaF,
						double twopit,
						double deltaT,
						double TwoDeltaToverN)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < dataSize)
	{
		
		__shared__ REAL8 ssum[MAX_THREADS];

		idx += lower; //accounts for the shift that was made in the original loop

		memset(ssum, 0, MAX_THREADS * sizeof(*ssum));

		int tid = threadIdx.x;
		int bid = blockIdx.x;

		REAL8 plainTemplateReal = FplusScaled * freqModelhPlus_Data[idx].re  
                          	          +  FcrossScaled * freqModelhCross_Data[idx].re;
		REAL8 plainTemplateImag = FplusScaled * freqModelhPlus_Data[idx].im
					  +  FcrossScaled * freqModelhCross_Data[idx].im;
		
                
		/* do time-shifting...             */
		/* (also un-do 1/deltaT scaling): */
		double f = ((double) idx) * deltaF;

		/* real & imag parts of  exp(-2*pi*i*f*deltaT): */
		double re = cos(twopit * f);
		double im = - sin(twopit * f);

		REAL8 templateReal = (plainTemplateReal*re - plainTemplateImag*im) / deltaT;
		REAL8 templateImag = (plainTemplateReal*im + plainTemplateImag*re) / deltaT;
		double dataReal     = freqData_Data[idx].re / deltaT;
		double dataImag     = freqData_Data[idx].im / deltaT;
		
		/* compute squared difference & 'chi-squared': */
		double diffRe       = dataReal - templateReal;         // Difference in real parts...
		double diffIm       = dataImag - templateImag;         // ...and imaginary parts, and...
				
		double diffSquared  = diffRe*diffRe + diffIm*diffIm ;  // ...squared difference of the 2 complex figures.
		
		ssum[tid] = ((TwoDeltaToverN * diffSquared) / oneSidedNoisePowerSpectrum_Data[idx]);

		/*****   REDUCTION    *****/
		
		__syncthreads(); //all the temps should have data before we add them up

		for (int i = blockDim.x / 2; i > 0; i >>= 1) { /* per block */
			if (tid < i)
			   ssum[tid] += ssum[tid + i];

			__syncthreads();
		}

		d_sum[bid] = ssum[0];
		
	}
}

__global__ void chisquared_LogLikelihood_Kernel_2(double *d_re, double *d_im, double deltaF, double twopit, int lower, int shift, double dataSize)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx < dataSize)
	{
		idx += lower;

		double f = ((double) idx) * deltaF;

		d_re[idx - lower + shift] = cos(twopit * f);
		d_im[idx - lower + shift] = - sin(twopit * f);
	}
}

__global__ void reduction_Kernel(REAL8 *d_temp, REAL8 *d_sum, int dataSize)
{
	__shared__ REAL8 ssum[MAX_THREADS];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	
	ssum[tid] = (idx < dataSize) ? d_temp[idx] : 0;

	__syncthreads();

	for (int i = blockDim.x / 2; i > 0; i >>= 1) { /* per block */
		if (tid < i)
		   ssum[tid] += ssum[tid + i];

		__syncthreads();
	}

	if (tid == 0) d_sum[bid] = ssum[0];
}

REAL8 LALInferenceUndecomposedFreqDomainLogLikelihood_GPU (LALInferenceVariables *currentParams, LALInferenceIFOData * data, 
                              LALInferenceTemplateFunction *_template)
/***************************************************************/
/* (log-) likelihood function.                                 */
/* Returns the non-normalised logarithmic likelihood.          */
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
/* Required (`currentParams') parameters are:                  */
/*   - "rightascension"  (REAL8, radian, 0 <= RA <= 2pi)       */
/*   - "declination"     (REAL8, radian, -pi/2 <= dec <=pi/2)  */
/*   - "polarisation"    (REAL8, radian, 0 <= psi <= ?)        */
/*   - "distance"        (REAL8, Mpc, >0)                      */
/*   - "time"            (REAL8, GPS sec.)                     */
/***************************************************************/
{
  //static int timeDomainWarning = 0;
  double Fplus, Fcross;
  double FplusScaled, FcrossScaled;
  double diffRe, diffIm, diffSquared;
  double dataReal, dataImag;
  REAL8 loglikeli;
  REAL8 plainTemplateReal, plainTemplateImag;
  REAL8 templateReal, templateImag;
  int i, lower, upper;
  LALInferenceIFOData *dataPtr;
  double ra, dec, psi, distMpc, gmst;
  double GPSdouble;
  LIGOTimeGPS GPSlal;
  double chisquared;
  double timedelay;  /* time delay b/w iterferometer & geocenter w.r.t. sky location */
  double timeshift;  /* time shift (not necessarily same as above)                   */
  double deltaT, TwoDeltaToverN, deltaF, twopit;
  double timeTmp;
  double mc;
  int different;
  UINT4 logDistFlag=0;
  LALStatus status;
  memset(&status,0,sizeof(status));
  LALInferenceVariables intrinsicParams;

  logDistFlag=LALInferenceCheckVariable(currentParams, "logdistance");
  if(LALInferenceCheckVariable(currentParams,"logmc")){
    mc=exp(*(REAL8 *)LALInferenceGetVariable(currentParams,"logmc"));
    LALInferenceAddVariable(currentParams,"chirpmass",&mc,LALINFERENCE_REAL8_t,LALINFERENCE_PARAM_OUTPUT);
  }

  /* determine source's sky location & orientation parameters: */
  ra        = *(REAL8*) LALInferenceGetVariable(currentParams, "rightascension"); /* radian      */
  dec       = *(REAL8*) LALInferenceGetVariable(currentParams, "declination");    /* radian      */
  psi       = *(REAL8*) LALInferenceGetVariable(currentParams, "polarisation");   /* radian      */
  GPSdouble = *(REAL8*) LALInferenceGetVariable(currentParams, "time");           /* GPS seconds */
	if(logDistFlag)
		 distMpc = exp(*(REAL8*)LALInferenceGetVariable(currentParams,"logdistance"));
	else
		 distMpc   = *(REAL8*) LALInferenceGetVariable(currentParams, "distance");       /* Mpc         */

  /* figure out GMST: */
  //XLALINT8NSToGPS(&GPSlal, floor(1e9 * GPSdouble + 0.5));
  XLALGPSSetREAL8(&GPSlal, GPSdouble);
  //UandA.units    = MST_RAD;
  //UandA.accuracy = LALLEAPSEC_LOOSE;
  //LALGPStoGMST1(&status, &gmst, &GPSlal, &UandA);
  gmst=XLALGreenwichMeanSiderealTime(&GPSlal);
  intrinsicParams.head      = NULL;
  intrinsicParams.dimension = 0;
  LALInferenceCopyVariables(currentParams, &intrinsicParams);
  LALInferenceRemoveVariable(&intrinsicParams, "rightascension");
  LALInferenceRemoveVariable(&intrinsicParams, "declination");
  LALInferenceRemoveVariable(&intrinsicParams, "polarisation");
  LALInferenceRemoveVariable(&intrinsicParams, "time");
	if(logDistFlag)
			LALInferenceRemoveVariable(&intrinsicParams, "logdistance");
	else
			LALInferenceRemoveVariable(&intrinsicParams, "distance");
  // TODO: add pointer to template function here.
  // (otherwise same parameters but different template will lead to no re-computation!!)

  chisquared = 0.0;
  /* loop over data (different interferometers): */
  dataPtr = data;

  //float totalTime;

  while (dataPtr != NULL) {
    /* The parameters the Likelihood function can handle by itself   */
    /* (and which shouldn't affect the template function) are        */
    /* sky location (ra, dec), polarisation and signal arrival time. */
    /* Note that the template function shifts the waveform to so that*/
	/* t_c corresponds to the "time" parameter in                    */
	/* IFOdata->modelParams (set, e.g., from the trigger value).     */
    
    /* Reset log-likelihood */
    dataPtr->loglikelihood = 0.0;

    /* Compare parameter values with parameter values corresponding  */
    /* to currently stored template; ignore "time" variable:         */
    if (LALInferenceCheckVariable(dataPtr->modelParams, "time")) {
      timeTmp = *(REAL8 *) LALInferenceGetVariable(dataPtr->modelParams, "time");
      LALInferenceRemoveVariable(dataPtr->modelParams, "time");
    }
    else timeTmp = GPSdouble;
    different = LALInferenceCompareVariables(dataPtr->modelParams, &intrinsicParams);
    /* "different" now may also mean that "dataPtr->modelParams" */
    /* wasn't allocated yet (as in the very 1st iteration).      */

    if (different) { /* template needs to be re-computed: */
      LALInferenceCopyVariables(&intrinsicParams, dataPtr->modelParams);
      LALInferenceAddVariable(dataPtr->modelParams, "time", &timeTmp, LALINFERENCE_REAL8_t,LALINFERENCE_PARAM_LINEAR);
      _template(dataPtr);
      if(XLALGetBaseErrno()==XLAL_FAILURE) /* Template generation failed in a known way, set -Inf likelihood */
          return(-DBL_MAX);

      if (dataPtr->modelDomain == LALINFERENCE_DOMAIN_TIME) {
//	if (!timeDomainWarning) {
//	  timeDomainWarning = 1;
//	  fprintf(stderr, "WARNING: using time domain template with frequency domain likelihood (in %s, line %d)\n", __FILE__, __LINE__);
//	}
        LALInferenceExecuteFT(dataPtr);
        /* note that the dataPtr->modelParams "time" element may have changed here!! */
        /* (during "template()" computation)  */
      }
    }
    else { /* no re-computation necessary. Return back "time" value, do nothing else: */
      LALInferenceAddVariable(dataPtr->modelParams, "time", &timeTmp, LALINFERENCE_REAL8_t,LALINFERENCE_PARAM_LINEAR);
    }

    /*-- Template is now in dataPtr->freqModelhPlus and dataPtr->freqModelhCross. --*/
    /*-- (Either freshly computed or inherited.)                            --*/

    /* determine beam pattern response (F_plus and F_cross) for given Ifo: */
    XLALComputeDetAMResponse(&Fplus, &Fcross,
                             dataPtr->detector->response,
			     ra, dec, psi, gmst);
    /* signal arrival time (relative to geocenter); */
    timedelay = XLALTimeDelayFromEarthCenter(dataPtr->detector->location,
                                             ra, dec, &GPSlal);
    /* (negative timedelay means signal arrives earlier at Ifo than at geocenter, etc.) */
    /* amount by which to time-shift template (not necessarily same as above "timedelay"): */
    timeshift =  (GPSdouble - (*(REAL8*) LALInferenceGetVariable(dataPtr->modelParams, "time"))) + timedelay;
    twopit    = LAL_TWOPI * timeshift;

    /* include distance (overall amplitude) effect in Fplus/Fcross: */
    FplusScaled  = Fplus  / distMpc;
    FcrossScaled = Fcross / distMpc;

    if (LALInferenceCheckVariable(currentParams, "crazyInjectionHLSign") &&
        *((INT4 *)LALInferenceGetVariable(currentParams, "crazyInjectionHLSign"))) {
      if (strstr(dataPtr->name, "H") || strstr(dataPtr->name, "L")) {
        FplusScaled *= -1.0;
        FcrossScaled *= -1.0;
      }
    }

    dataPtr->fPlus = FplusScaled;
    dataPtr->fCross = FcrossScaled;
    dataPtr->timeshift = timeshift;

    //FILE *testout=fopen("test_likeliLAL.txt","w");
    //fprintf(testout, "f PSD dataRe dataIm signalRe signalIm\n");
    /* determine frequency range & loop over frequency bins: */
    deltaT = dataPtr->timeData->deltaT;
    deltaF = 1.0 / (((double)dataPtr->timeData->data->length) * deltaT);
    // printf("deltaF %g, Nt %d, deltaT %g\n", deltaF, dataPtr->timeData->data->length, dataPtr->timeData->deltaT);
    lower = (UINT4)ceil(dataPtr->fLow / deltaF);
    upper = (UINT4)floor(dataPtr->fHigh / deltaF);
    TwoDeltaToverN = 2.0 * deltaT / ((double) dataPtr->timeData->data->length);

    /*****    CUDA  SUBSTITUTION  HERE    *****/

    int dataSize = upper-lower + 1;

    int numThreadsPerBlock = 0;

    int remainder[9]; //hold the remainders of the modulus operations
    int j = 0;
    //calculate the number of threads needed for this particular dataSize
    for (i = MAX_THREADS; i > 0; i >>= 1)
    {
        if (dataSize % i == 0) //split up the number of threads evenly across the dataset
        {
            numThreadsPerBlock = i;  
            break;
        }

        remainder[j] = dataSize % i; //store the remainder
        j++;

    }

    int numBlocks = dataSize / numThreadsPerBlock;

    //recalculate a better number of threads per block if the number of blocks is exceeding hardware limits
    if (numBlocks > 65000 || (numBlocks > 65000 && numThreadsPerBlock == 1))
    {
        int tMin = remainder[0]; //initialize the minimum remainder
        int tracker = 0; //tracker for which index had the minimum remainder
        for (j = 1; j < 9; j++)
        {
            if (remainder[j] < tMin && remainder[j] > 0)
            {
                tMin = remainder[j];
                tracker = j;
            }
        }
	
        switch (tracker)
        {
	    //choose the number of threads per block based on the least remainder
            case 0: numThreadsPerBlock = 512; break;
            case 1: numThreadsPerBlock = 256; break;
            case 2: numThreadsPerBlock = 128; break;
            case 3: numThreadsPerBlock = 64; break;
            case 4: numThreadsPerBlock = 32; break;
            case 5: numThreadsPerBlock = 16; break;
            case 6: numThreadsPerBlock = 8; break;
            case 7: numThreadsPerBlock = 4; break;
            default: numThreadsPerBlock = 2; break;
        }

        numBlocks = dataSize / numThreadsPerBlock + 1; //we need to give it an extra so it accounts for the data not evenly distributed
    }

    dim3 dimGrid(numBlocks); 
    dim3 dimBlock(numThreadsPerBlock);

    size_t memSize = dataSize * sizeof(double);

    //for concurrent streams
    //hipHostMalloc of host data
    //hipMalloc of device data
    //make two streams hipStream_t stream1 hipStream_t stream2
    //create the streams hipStreamCreate(&stream1)  hipStreamCreate(&stream2)
    //create event handlers  hipEvent_t start_event, stop_event;
    //create the event hipEventCreate(&start_event) hipEventCreate(&stop_event);
    //start recording the event hipEventRecord(start_event, 0);
    //launch kernel to streams kernel<<<dimGrid,dimBlocks,0,stream1>>>(args);  kernel<<<dimGrid,dimBlocks,0,stream1>>>(args);
    //optional: after each launch you need an event hipEventRecord(kernelEvent[i], streams[i])
    //copy the data hipMemcpyAsync(host, device, sizeof(memorySize), hipMemcpyDeviceToHost, stream1/2)
    //record the event hipEventRecord(stop_event, 0)
    //hipEventSynchronize(stop_event)
    //hipStreamDestroy(stream1/2);
    //hipEventDestroy(start_event);
    //hipEventDestroy(stop_event);
    //hipHostFree(host);
    //hipFree(device);

    double *d_re;
    double *d_im;

    double *h_re;
    double *h_im;

    //h_re = (double *)malloc(memSize);
    //h_im = (double *)malloc(memSize);

    hipHostMalloc((void **)&h_re, memSize);
    hipHostMalloc((void **)&h_im, memSize);

    hipMalloc((void **)&d_re, memSize);
    hipMalloc((void **)&d_im, memSize);

    hipStream_t stream1;
    hipStream_t stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    hipEventRecord(start_event, 0);

    chisquared_LogLikelihood_Kernel_2<<<dimGrid, dimBlock, 0, stream1>>>(d_re, d_im, deltaF, twopit, lower, 0, dataSize / 2);

    chisquared_LogLikelihood_Kernel_2<<<dimGrid, dimBlock, 0, stream2>>>(d_re, d_im, deltaF, twopit, lower + (dataSize / 2), dataSize / 2, dataSize / 2);

    // block until the device has completed
    //hipDeviceSynchronize();

    //checkCUDAError("kernel execution");

    hipMemcpyAsync(h_re, d_re, memSize, hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(h_im, d_im, memSize, hipMemcpyDeviceToHost, stream2);

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    //checkCUDAError("hipMemcpy");

    for (i=lower; i<=upper; ++i){
      /* derive template (involving location/orientation parameters) from given plus/cross waveforms: */
      plainTemplateReal = FplusScaled * dataPtr->freqModelhPlus->data->data[i].re  
                          +  FcrossScaled * dataPtr->freqModelhCross->data->data[i].re;
      plainTemplateImag = FplusScaled * dataPtr->freqModelhPlus->data->data[i].im  
                          +  FcrossScaled * dataPtr->freqModelhCross->data->data[i].im;

      /* real & imag parts of  exp(-2*pi*i*f*deltaT): */
      templateReal = (plainTemplateReal*h_re[i - lower] - plainTemplateImag*h_im[i - lower]) / deltaT;
      templateImag = (plainTemplateReal*h_im[i - lower] + plainTemplateImag*h_re[i - lower]) / deltaT;
      dataReal     = dataPtr->freqData->data->data[i].re / deltaT;
      dataImag     = dataPtr->freqData->data->data[i].im / deltaT;
      /* compute squared difference & 'chi-squared': */
      diffRe       = dataReal - templateReal;         // Difference in real parts...
      diffIm       = dataImag - templateImag;         // ...and imaginary parts, and...
      diffSquared  = diffRe*diffRe + diffIm*diffIm ;  // ...squared difference of the 2 complex figures.
      REAL8 temp = ((TwoDeltaToverN * diffSquared) / dataPtr->oneSidedNoisePowerSpectrum->data->data[i]);
      
      chisquared  += temp;
      dataPtr->loglikelihood -= temp;
    }
    
    hipFree(d_re);
    hipFree(d_im);
    hipHostFree(h_re);
    hipHostFree(h_im);

    //free(h_re);
    //free(h_im);

    //size_t sumMemSize = numBlocks * sizeof(REAL8);
    //size_t complexMemSize = (upper + 1) * sizeof(COMPLEX16); 
    //size_t realMemSize = (upper + 1) * sizeof(REAL8);

    //the host pointer to be used for the overall summation in host
    //REAL8 *h_sum;   

    //h_sum = (REAL8 *)malloc(sumMemSize);
    //memset(h_sum, 0, sumMemSize);

    //the device pointer for the per-block reduction summation
    //REAL8 *d_sum;   

    //device pointers for data from dataPtr
    //COMPLEX16 *d_freqModelhPlus_data;
    //COMPLEX16 *d_freqModelhCross_data;
    //COMPLEX16 *d_freqData_data;
    //REAL8 *d_oneSidedNoisePowerSpectrum_data;

    //device memory allocation
    //hipMalloc((void **)&d_sum, sumMemSize);
   
    //hipMalloc((void **)&d_freqModelhPlus_data, complexMemSize);
    //hipMalloc((void **)&d_freqModelhCross_data, complexMemSize);
    //hipMalloc((void **)&d_freqData_data, complexMemSize);
    //hipMalloc((void **)&d_oneSidedNoisePowerSpectrum_data, realMemSize);

    //hipMemcpy(d_freqModelhPlus_data, dataPtr->freqModelhPlus->data->data, complexMemSize, hipMemcpyHostToDevice);
    //hipMemcpy(d_freqModelhCross_data, dataPtr->freqModelhCross->data->data, complexMemSize, hipMemcpyHostToDevice);
    //hipMemcpy(d_freqData_data, dataPtr->freqData->data->data, complexMemSize, hipMemcpyHostToDevice);
    //hipMemcpy(d_oneSidedNoisePowerSpectrum_data, dataPtr->oneSidedNoisePowerSpectrum->data->data, realMemSize, hipMemcpyHostToDevice);
    
    //hipEventRecord(start, 0);

    //launch the kernel
    //chisquared_LogLikelihood_Kernel<<<dimGrid, dimBlock>>>(d_sum, lower, dataSize,
							//d_freqModelhPlus_data,
							//d_freqModelhCross_data,
							//d_freqData_data,
							//d_oneSidedNoisePowerSpectrum_data,
	    				   		//FplusScaled,
            				   		//FcrossScaled,
            				   		//deltaF,
            				   		//twopit,
            				   		//deltaT,
            				   		//TwoDeltaToverN);

    //chisquared_LogLikelihood_Kernel<<<dimGrid, dimBlock>>>(d_re, d_im);

    // block until the device has completed
    //hipDeviceSynchronize();

    // check if kernel execution generated an error
    //checkCUDAError("kernel execution");

    //hipEventRecord(stop, 0);
    //hipEventSynchronize(stop);

    //hipMemcpy(h_sum, d_sum, sumMemSize, hipMemcpyDeviceToHost);

    //checkCUDAError("hipMemcpy");

    //for (i = 0; i < numBlocks; i++)
    //{
    //    chisquared += h_sum[i];
    //    dataPtr->loglikelihood -= h_sum[i];
    //}
    
    //hipFree(d_sum);
    //hipFree(d_freqModelhPlus_data);
    //hipFree(d_freqModelhCross_data);
    //hipFree(d_freqData_data);
    //hipFree(d_oneSidedNoisePowerSpectrum_data);
    //free(h_sum);

    //hipEventElapsedTime(&time, start, stop);
    //printf ("Time for the kernel: %f ms\n", time);
    //totalTime += time;

    dataPtr = dataPtr->next;
	//fclose(testout);
  }
  //printf ("Total Time for kernel and loop: %f ms\n", totalTime);
  loglikeli = -1.0 * chisquared; // note (again): the log-likelihood is unnormalised!
  LALInferenceDestroyVariables(&intrinsicParams);
  return(loglikeli);
}

