#include <lal/LALDatatypes.h>
#include "CudaFunctions.h"

void XLALCudaError(hipError_t error, const char *file, int line)
{
    if(error != hipSuccess)	
    {	   
        fprintf( stderr, "%s:%d %s\n", file, line, hipGetErrorString(error));
        exit(1);
    }
}

void XLALCudaFFTError(hipfftResult_t error, const char *file, int line)
{
    if(error != HIPFFT_SUCCESS) 
    {
        fprintf( stderr, "%s:%d Cuda FFT Error: %d\n", file, line, error);
        exit(1);
    }
}

int cudafft_execute_r2c(hipfftHandle plan,
    hipfftComplex *output, const hipfftReal *input,
    hipfftComplex *d_output, hipfftReal *d_input,UINT4 size)
{
    UINT4 inputBytes = size * sizeof(hipfftReal);
    UINT4 outputBytes = (size/2 + 1) * sizeof(hipfftComplex);

    XLALCUDACHECK(hipMemcpy( d_input, input, inputBytes, hipMemcpyHostToDevice ));

    XLALCUDAFFTCHECK(hipfftExecR2C(plan, d_input, d_output));

    XLALCUDACHECK(hipMemcpy( output, d_output, outputBytes, hipMemcpyDeviceToHost ));

    return 0;
}

int cudafft_execute_c2r(hipfftHandle plan,
    hipfftReal *output, const hipfftComplex *input,
    hipfftReal *d_output, hipfftComplex *d_input, UINT4 size)
{
    UINT4 inputBytes = (size/2 + 1) * sizeof(hipfftComplex);
    UINT4 outputBytes = size * sizeof(hipfftReal);

    XLALCUDACHECK(hipMemcpy( d_input, input, inputBytes, hipMemcpyHostToDevice ));

    XLALCUDAFFTCHECK(hipfftExecC2R(plan, d_input, d_output));

    XLALCUDACHECK(hipMemcpy( output, d_output, outputBytes, hipMemcpyDeviceToHost ));

    return 0;
}

int cudafft_execute_c2c(hipfftHandle plan,
    hipfftComplex *output, const hipfftComplex *input,
    hipfftComplex *d_output, hipfftComplex *d_input,
    INT4 direction, UINT4 size)
{
    UINT4 nBytes = size * sizeof(hipfftComplex);

    XLALCUDACHECK(hipMemcpy( d_input, input, nBytes, hipMemcpyHostToDevice ));

    XLALCUDAFFTCHECK(hipfftExecC2C(plan, d_input, d_output, direction));

    XLALCUDACHECK(hipMemcpy( output, d_output, nBytes, hipMemcpyDeviceToHost ));

    return 0;
}
