#include <lal/LALDatatypes.h>
#include "CudaFunctions.h"

int cudafft_execute_r2c(hipfftHandle plan,
    hipfftComplex *output, const hipfftReal *input,
    hipfftComplex *d_output, hipfftReal *d_input,UINT4 size)
{
    UINT4 inputBytes = size * sizeof(hipfftReal);
    UINT4 outputBytes = (size/2 + 1) * sizeof(hipfftComplex);

    hipMemcpy( d_input, input, inputBytes, hipMemcpyHostToDevice );

    hipfftExecR2C(plan, d_input, d_output);

    hipMemcpy( output, d_output, outputBytes, hipMemcpyDeviceToHost );

    return 0;
}

int cudafft_execute_c2r(hipfftHandle plan,
    hipfftReal *output, const hipfftComplex *input,
    hipfftReal *d_output, hipfftComplex *d_input, UINT4 size)
{
    UINT4 inputBytes = (size/2 + 1) * sizeof(hipfftComplex);
    UINT4 outputBytes = size * sizeof(hipfftReal);

    hipMemcpy( d_input, input, inputBytes, hipMemcpyHostToDevice );

    hipfftExecC2R(plan, d_input, d_output);

    hipMemcpy( output, d_output, outputBytes, hipMemcpyDeviceToHost );

    return 0;
}

int cudafft_execute_c2c(hipfftHandle plan,
    hipfftComplex *output, const hipfftComplex *input,
    hipfftComplex *d_output, hipfftComplex *d_input,
    INT4 direction, UINT4 size)
{
    UINT4 nBytes = size * sizeof(hipfftComplex);

    hipMemcpy( d_input, input, nBytes, hipMemcpyHostToDevice );

    hipfftExecC2C(plan, d_input, d_output, direction);

    hipMemcpy( output, d_output, nBytes, hipMemcpyDeviceToHost );

    return 0;
}
