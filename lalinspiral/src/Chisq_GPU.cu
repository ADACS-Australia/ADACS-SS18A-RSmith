#include "hip/hip_runtime.h"
/*
*  Copyright (C) 2010 Karsten Wiesner
*
*  This program is free software; you can redistribute it and/or modify
*  it under the terms of the GNU General Public License as published by
*  the Free Software Foundation; either version 2 of the License, or
*  (at your option) any later version.
*
*  This program is distributed in the hope that it will be useful,
*  but WITHOUT ANY WARRANTY; without even the implied warranty of
*  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*  GNU General Public License for more details.
*
*  You should have received a copy of the GNU General Public License
*  along with with program; see the file COPYING. If not, write to the
*  Free Software Foundation, Inc., 59 Temple Place, Suite 330, Boston,
*  MA  02111-1307  USA
*/

/*-----------------------------------------------------------------------
 *
 * File Name: Chisq_GPU.cu
 *
 * Author: Wiesner, K.
 *
 *
 *-----------------------------------------------------------------------
 */

#include <stdio.h>
#include <hipfft/hipfft.h>
#define LAL_USE_OLD_COMPLEX_STRUCTS
#include <lal/LALAtomicDatatypes.h>

static void CudaError(hipError_t error, const char *file, int line)
{
        if(error != hipSuccess)
        {
                printf("%s:%d %s\n", file, line, hipGetErrorString(error));
                exit(-1);
        }
}

#define CUDACHECK(e) (CudaError(e, __FILE__, __LINE__))

#define NUM_THREADS_MAX 512

////////////////////////////////////////////////////////////////////////////////
//! Chi Squared test kernel
//! @param g_chisq    output data in global memory
//! @param g_q        input data in global memory
//! @param g_data     time series (output of IFFT) in global memory
//! @param chisqNorm  normalization constant
////////////////////////////////////////////////////////////////////////////////
__global__ void
chisqKernel( REAL4* g_chisq, COMPLEX8* g_q, COMPLEX8 *g_data, 
	     UINT4 numPoints, UINT4 numChisqBins, REAL4 chisqNorm) 
{
  for (unsigned l=0; l < numChisqBins; l++)
    {
      unsigned j= blockIdx.x * blockDim.x + threadIdx.x;
      
      REAL4 Xl = g_data[l*numPoints + j].re;
      REAL4 Yl = g_data[l*numPoints + j].im;
      
      REAL4 deltaXl = chisqNorm * Xl -
	(chisqNorm * g_q[j].re / (REAL4) (numChisqBins));
      REAL4 deltaYl = chisqNorm * Yl -
	(chisqNorm * g_q[j].im / (REAL4) (numChisqBins));
      
      g_chisq[j] += deltaXl * deltaXl + deltaYl * deltaYl;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Chi Squared test lalinspiral interface
////////////////////////////////////////////////////////////////////////////////
extern "C"
void Chisq_GPU (REAL4* chisq, COMPLEX8* q, COMPLEX8* qtilde, UINT4* chisqBin,
		UINT4 numPoints, UINT4 numChisqBins, REAL4 chisqNorm )
{
  
  // hipSetDevice( 0 ) already done by cufft.

  COMPLEX8* d_q;        // input snr timeseries
  CUDACHECK( hipMalloc( (void**) &d_q, numPoints * sizeof(COMPLEX8)) );
  //printf("Allocated device d_q\n");
  CUDACHECK( hipMemcpy( d_q, q, numPoints * sizeof(COMPLEX8), hipMemcpyHostToDevice) );
  //printf("Memcopy q to device d_q done\n");

  REAL4*    d_chisq;    // output chisq calculation
  CUDACHECK( hipMalloc( (void**) &d_chisq, numPoints * sizeof(REAL4)) );
  CUDACHECK( hipMemset( d_chisq, 0, numPoints  * sizeof(REAL4) ));
  //printf("Allocated and zero initialized device d_chisq\n");

  // d_data is the output from the batch FFT, and input
  // for the Chisq calculations
  COMPLEX8* d_data;
  CUDACHECK( hipMalloc( (void **)&d_data, numPoints * numChisqBins * sizeof(COMPLEX8)) );
  CUDACHECK( hipMemset( d_data, 0, numPoints * numChisqBins * sizeof(COMPLEX8) ));  
  //printf("Allocated and zero initialized device d_data\n");

  COMPLEX8 *d_qtildeBin;	
  CUDACHECK(hipMalloc( (void **)&d_qtildeBin, numPoints * numChisqBins * sizeof(COMPLEX8)) );
  CUDACHECK( hipMemset( d_qtildeBin, 0, numPoints * numChisqBins * sizeof(COMPLEX8) ));
  //printf("Allocated and zero initialized device q_tildeBin\n");

  //printf("\nCopy portions of qtilde to bins of zeroinitialized d_qtildeBin device memory:\n");
  //printf("d_qtildeBin (dest) points to =%p\n", d_qtildeBin);
  //printf("qtilde (src) points to = %p\n", qtilde);
  
  for( unsigned i = 0; i < numChisqBins; i++ )
    {
      //printf ("Copy bin #%d\n", i);
      //printf("dest=%p\n", (d_qtildeBin + i * numPoints + chisqBin[i]));
      //printf("src=%p   %p    0x%lx      \n", (qtilde + chisqBin[i]), qtilde, chisqBin[i] * sizeof(COMPLEX8));
      //printf("num=0x%lx\n", (chisqBin[i+1] - chisqBin[i]) * sizeof(COMPLEX8));
      
      CUDACHECK( hipMemcpy( &d_qtildeBin[i*numPoints] + chisqBin[i],
			     (qtilde + chisqBin[i]),
			     (chisqBin[i+1] - chisqBin[i]) * sizeof(COMPLEX8),      
			     hipMemcpyHostToDevice) );
    }
 
  //printf("\nStarting batch executions of %d cuda IFFTs -----------------------------\n", numChisqBins);
  
  hipfftHandle batchPlan;
  hipfftPlan1d( &batchPlan, numPoints, HIPFFT_C2C, numChisqBins );
  
  hipEvent_t start, stop;
  CUDACHECK(hipEventCreate(&start));
  CUDACHECK(hipEventCreate(&stop));
  CUDACHECK(hipEventRecord(start, 0));
	
  hipfftExecC2C( batchPlan, 
		(hipfftComplex *)d_qtildeBin, (hipfftComplex *)d_data, 
		HIPFFT_BACKWARD );
		
  hipfftDestroy(batchPlan);	

  // GPU chisq calculation ----------------------------------------------------------
  unsigned numThreadsX = (unsigned) NUM_THREADS_MAX;
  unsigned numBlocksX = numPoints / numThreadsX;

  dim3  grid( numBlocksX, 1, 1);
  dim3  threads( numThreadsX, 1, 1);

  //printf("\nStarting grid of kernels: %d blocks with %d threads ----------------\n", numBlocksX, numThreadsX);	
  
  chisqKernel<<< grid, threads >>>( d_chisq, d_q, d_data, numPoints, numChisqBins, chisqNorm );
  
  hipDeviceSynchronize(); // implicit in cudaPrintfDisplay
  // cudaPrintfDisplay(stdout, true);
  
  // copy result from device to host ----------------------------------------
  CUDACHECK( hipMemcpy( chisq, d_chisq, numPoints * sizeof(REAL4), hipMemcpyDeviceToHost) );

  // cleanup device memory
  CUDACHECK(hipFree(d_q));
  CUDACHECK(hipFree(d_chisq));
  CUDACHECK(hipFree(d_data));
  CUDACHECK(hipFree(d_qtildeBin));

}
